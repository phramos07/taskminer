#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <openacc.h>
#define IPMACC_MAX1(A)   (A)
#define IPMACC_MAX2(A,B) (A>B?A:B)
#define IPMACC_MAX3(A,B,C) (A>B?(A>C?A:(B>C?B:C)):(B>C?C:B))
#ifdef __cplusplus
#include "openacc_container.h"
#endif

#include <hip/hip_runtime.h>





#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

int SIZE;

float *a;
float *b;
float *c;

FILE *fil;
FILE *out;





  __global__ void __generated_kernel_region_0(float * a,float * c,int  s,float * b);
 
void GPU__main__mul_CPU__init(int s)
{
  int i, j;
  long long int AI1 [6];
  AI1 [0] = s > 0;
  AI1 [1] = (AI1 [0] ? s : 0);
  AI1 [2] = s * AI1 [1];
  AI1 [3] = AI1 [2] + s;
  AI1 [4] = AI1 [3] * 4;
  AI1 [5] = AI1 [4] / 4;
  

	ipmacc_prompt((char*)"IPMACC: memory allocation a\n");
acc_create((void*)a,(AI1[5]+0)*sizeof(float ));
ipmacc_prompt((char*)"IPMACC: memory allocation c\n");
acc_create((void*)c,(AI1[5]+0)*sizeof(float ));
ipmacc_prompt((char*)"IPMACC: memory allocation b\n");
acc_create((void*)b,(AI1[5]+0)*sizeof(float ));
	ipmacc_prompt((char*)"IPMACC: memory copyin a\n");
acc_copyin((void*)a,(AI1[5]+0)*sizeof(float ));
ipmacc_prompt((char*)"IPMACC: memory copyin c\n");
acc_copyin((void*)c,(AI1[5]+0)*sizeof(float ));
ipmacc_prompt((char*)"IPMACC: memory copyin b\n");
acc_copyin((void*)b,(AI1[5]+0)*sizeof(float ));


{


  


/* kernel call statement [0, -1]*/
{
if (getenv("IPMACC_VERBOSE")) printf("IPMACC: Launching kernel 0 > gridDim: %d\tblockDim: %d\n",(((abs((int)((s))-(0+0)))/(1)))/256+(((((abs((int)((s))-(0+0)))/(1)))%(256))==0?0:1),256);
__generated_kernel_region_0<<<(((abs((int)((s))-(0+0)))/(1)))/256+(((((abs((int)((s))-(0+0)))/(1)))%(256))==0?0:1),256>>>(
(float *)acc_deviceptr((void*)a),
(float *)acc_deviceptr((void*)c),
s,
(float *)acc_deviceptr((void*)b));
}
/* kernel call statement*/
if (getenv("IPMACC_VERBOSE")) printf("IPMACC: Synchronizing the region with host\n");
{
hipError_t err=hipDeviceSynchronize();
if(err!=hipSuccess){
printf("Kernel Launch Error! error code (%d)\n",err);
assert(0&&"Launch Failure!\n");}
}



}
	ipmacc_prompt((char*)"IPMACC: memory copyout a\n");
acc_copyout_and_keep((void*)a,(AI1[5]+0)*sizeof(float ));
ipmacc_prompt((char*)"IPMACC: memory copyout c\n");
acc_copyout_and_keep((void*)c,(AI1[5]+0)*sizeof(float ));
ipmacc_prompt((char*)"IPMACC: memory copyout b\n");
acc_copyout_and_keep((void*)b,(AI1[5]+0)*sizeof(float ));



}


void CPU__main__mul_CPU__init(int s)
{
  int i, j;
  for (i = 0; i < s; ++i) {
    for (j = 0; j < s; ++j) {
      a [i * s + j] = (float)i + j % 100;
      b [i * s + j] = (float)i + j % 100;
      c [i * s + j] = 0.0f;
    }
  }
}

void init(int s)
{
  int i, j;
  for (i = 0; i < s; ++i) {
    for (j = 0; j < s; ++j) {
      a [i * s + j] = (float)i + j % 100;
      b [i * s + j] = (float)i + j % 100;
      c [i * s + j] = 0.0f;
    }
  }
}


void print(int s)
{
  int i, j;
  for (i = 0; i < s; ++i) {
    for (j = 0; j < s; ++j) {
      fprintf(out, "%f ", c [i * s + j]);
    }
    fprintf(out, "\n");
  }
}






  __global__ void __generated_kernel_region_1(float * a,float * b,float * c,int  s,float  sum);
 
void GPU__main__mul_CPU(int s)
{
  a = (float*)malloc(sizeof(float) * SIZE * SIZE);
  b = (float*)malloc(sizeof(float) * SIZE * SIZE);
  c = (float*)malloc(sizeof(float) * SIZE * SIZE);

  GPU__main__mul_CPU__init(SIZE);

  int i, j, k;
  float sum = 0.0;
  float start, finish, elapsed;
  start = (float)clock() / (CLOCKS_PER_SEC * 1000);
  long long int AI1 [10];
  AI1 [0] = s > 0;
  AI1 [1] = (AI1 [0] ? s : 0);
  AI1 [2] = s * AI1 [1];
  AI1 [3] = AI1 [2] + s;
  AI1 [4] = AI1 [3] * 4;
  AI1 [5] = AI1 [4] / 4;
  AI1 [6] = s * s;
  AI1 [7] = s + AI1 [6];
  AI1 [8] = AI1 [7] * 4;
  AI1 [9] = AI1 [8] / 4;
  

	ipmacc_prompt((char*)"IPMACC: memory allocation a\n");
acc_create((void*)a,(AI1[5]+0)*sizeof(float ));
ipmacc_prompt((char*)"IPMACC: memory allocation c\n");
acc_create((void*)c,(AI1[5]+0)*sizeof(float ));
ipmacc_prompt((char*)"IPMACC: memory allocation b\n");
acc_create((void*)b,(AI1[9]+0)*sizeof(float ));
	ipmacc_prompt((char*)"IPMACC: memory copyin a\n");
acc_copyin((void*)a,(AI1[5]+0)*sizeof(float ));
ipmacc_prompt((char*)"IPMACC: memory copyin c\n");
acc_copyin((void*)c,(AI1[5]+0)*sizeof(float ));
ipmacc_prompt((char*)"IPMACC: memory copyin b\n");
acc_copyin((void*)b,(AI1[9]+0)*sizeof(float ));


{


  


/* kernel call statement [1, -1]*/
{
if (getenv("IPMACC_VERBOSE")) printf("IPMACC: Launching kernel 1 > gridDim: %d\tblockDim: %d\n",(((abs((int)((s))-(0+0)))/(1)))/256+(((((abs((int)((s))-(0+0)))/(1)))%(256))==0?0:1),256);
__generated_kernel_region_1<<<(((abs((int)((s))-(0+0)))/(1)))/256+(((((abs((int)((s))-(0+0)))/(1)))%(256))==0?0:1),256>>>(
(float *)acc_deviceptr((void*)a),
(float *)acc_deviceptr((void*)b),
(float *)acc_deviceptr((void*)c),
s,
sum);
}
/* kernel call statement*/
if (getenv("IPMACC_VERBOSE")) printf("IPMACC: Synchronizing the region with host\n");
{
hipError_t err=hipDeviceSynchronize();
if(err!=hipSuccess){
printf("Kernel Launch Error! error code (%d)\n",err);
assert(0&&"Launch Failure!\n");}
}



}
	ipmacc_prompt((char*)"IPMACC: memory copyout a\n");
acc_copyout_and_keep((void*)a,(AI1[5]+0)*sizeof(float ));
ipmacc_prompt((char*)"IPMACC: memory copyout c\n");
acc_copyout_and_keep((void*)c,(AI1[5]+0)*sizeof(float ));
ipmacc_prompt((char*)"IPMACC: memory copyout b\n");
acc_copyout_and_keep((void*)b,(AI1[9]+0)*sizeof(float ));



  finish = (float)clock() / (CLOCKS_PER_SEC * 1000);
  elapsed = finish - start;
  fprintf(fil, "%.10lf,", elapsed);

  
  free(a);
  free(b);
  free(c);
}


void CPU__main__mul_CPU(int s)
{
  a = (float*)malloc(sizeof(float) * SIZE * SIZE);
  b = (float*)malloc(sizeof(float) * SIZE * SIZE);
  c = (float*)malloc(sizeof(float) * SIZE * SIZE);

  GPU__main__mul_CPU__init(SIZE);

  int i, j, k;
  float sum = 0.0;
  float start, finish, elapsed;
  start = (float)clock() / (CLOCKS_PER_SEC * 1000);
  for (i = 0; i < s; ++i) {
    for (j = 0; j < s; ++j) {
      sum = 0.0;
      for (k = 0; k < s; ++k) {
        sum = sum + a [i * s + k] * b [k * s + j];
      }
      c [i * s + j] = sum;
    }
  }
  finish = (float)clock() / (CLOCKS_PER_SEC * 1000);
  elapsed = finish - start;
  fprintf(fil, "%.10lf,", elapsed);

  
  free(a);
  free(b);
  free(c);
}

void mul_CPU(int s)
{
  a = (float*)malloc(sizeof(float) * SIZE * SIZE);
  b = (float*)malloc(sizeof(float) * SIZE * SIZE);
  c = (float*)malloc(sizeof(float) * SIZE * SIZE);

  init(SIZE);

  int i, j, k;
  float sum = 0.0;
  float start, finish, elapsed;
  start = (float)clock() / (CLOCKS_PER_SEC * 1000);
  for (i = 0; i < s; ++i) {
    for (j = 0; j < s; ++j) {
      sum = 0.0;
      for (k = 0; k < s; ++k) {
        sum = sum + a [i * s + k] * b [k * s + j];
      }
      c [i * s + j] = sum;
    }
  }
  finish = (float)clock() / (CLOCKS_PER_SEC * 1000);
  elapsed = finish - start;
  fprintf(fil, "%.10lf,", elapsed);

  
  free(a);
  free(b);
  free(c);
}


int GPU__main(int argc, char *argv[])
{
  if (argc != 2) {
    return 1;
  }
  SIZE = atoi(argv [1]);

  fil = fopen("time_cpu.csv", "w+");
  out = fopen("result_cpu.txt", "w+");

  fprintf(fil, "SIZE,matrix multiplication CPU,\n");

  fprintf(fil, "%d,", SIZE);
  GPU__main__mul_CPU(SIZE);
  fprintf(fil, "\n");

  fclose(fil);
  fclose(out);
  return 0;
}

int main(int argc, char *argv[])
{
  if (argc != 2) {
    return 1;
  }
  SIZE = atoi(argv [1]);

  fil = fopen("time_cpu.csv", "w+");
  out = fopen("result_cpu.txt", "w+");

  fprintf(fil, "SIZE,matrix multiplication CPU,\n");

  fprintf(fil, "%d,", SIZE);
  GPU__main__mul_CPU(SIZE);
  fprintf(fil, "\n");

  fclose(fil);
  fclose(out);
  return 0;
}



 __global__ void __generated_kernel_region_0(float * a,float * c,int  s,float * b){
int __kernel_getuid_x=threadIdx.x+blockIdx.x*blockDim.x;
int __kernel_getuid_y=threadIdx.y+blockIdx.y*blockDim.y;
int __kernel_getuid_z=threadIdx.z+blockIdx.z*blockDim.z;
int  i;
int  j;
{
{
{
 i=0+(__kernel_getuid_x);
if( i < s)
{
for(j = 0; j < s; ++j)
{
      a [i * s + j] = (float)i + j % 100;
      b [i * s + j] = (float)i + j % 100;
      c [i * s + j] = 0.0f;
    }
}

}
}
}
//append writeback of scalar variables
}

 __global__ void __generated_kernel_region_1(float * a,float * b,float * c,int  s,float  sum){
int __kernel_getuid_x=threadIdx.x+blockIdx.x*blockDim.x;
int __kernel_getuid_y=threadIdx.y+blockIdx.y*blockDim.y;
int __kernel_getuid_z=threadIdx.z+blockIdx.z*blockDim.z;
int  i;
int  k;
int  j;
{
{
{
 i=0+(__kernel_getuid_x);
if( i < s)
{
for(j = 0; j < s; ++j)
{
      sum = 0.0;
for(k = 0; k < s; ++k)
{
        sum = sum + a [i * s + k] * b [k * s + j];
      }
c [i * s + j] = sum;
    }
}

}
}
}
//append writeback of scalar variables
}

