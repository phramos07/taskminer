#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <openacc.h>
#define IPMACC_MAX1(A)   (A)
#define IPMACC_MAX2(A,B) (A>B?A:B)
#define IPMACC_MAX3(A,B,C) (A>B?(A>C?A:(B>C?B:C)):(B>C?C:B))
#ifdef __cplusplus
#include "openacc_container.h"
#endif

#include <hip/hip_runtime.h>



#include <unistd.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdarg.h>
#include <string.h>


#include "../../common/polybenchUtilFuncts.h"


#define PERCENT_DIFF_ERROR_THRESHOLD 0.05


#define NI 512
#define NJ 512
#define NK 512


#define ALPHA 32412.0f
#define BETA 2123.0f


typedef float DATA_TYPE;

#define GPU_DEVICE 1

void CPU__gemm(DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *C)
{
  int i, j, k;

  for (i = 0; i < NI; i++) {
    for (j = 0; j < NJ; j++) {
      C [i * NJ + j] *= BETA;

      for (k = 0; k < NK; ++k) {
        C [i * NJ + j] += ALPHA * A [i * NK + k] * B [k * NJ + j];
      }
    }
  }
}

  __global__ void __generated_kernel_region_0(DATA_TYPE * A,DATA_TYPE * B,DATA_TYPE * C);
 
void GPU__gemm(DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *C)
{
  int i, j, k;

  

	ipmacc_prompt((char*)"IPMACC: memory allocation A\n");
acc_present_or_create((void*)A,(262143+0)*sizeof(DATA_TYPE ));
ipmacc_prompt((char*)"IPMACC: memory allocation B\n");
acc_present_or_create((void*)B,(262143+0)*sizeof(DATA_TYPE ));
ipmacc_prompt((char*)"IPMACC: memory allocation C\n");
acc_present_or_create((void*)C,(262143+0)*sizeof(DATA_TYPE ));
	ipmacc_prompt((char*)"IPMACC: memory copyin A\n");
acc_pcopyin((void*)A,(262143+0)*sizeof(DATA_TYPE ));
ipmacc_prompt((char*)"IPMACC: memory copyin B\n");
acc_pcopyin((void*)B,(262143+0)*sizeof(DATA_TYPE ));
ipmacc_prompt((char*)"IPMACC: memory copyin C\n");
acc_pcopyin((void*)C,(262143+0)*sizeof(DATA_TYPE ));


{


  


/* kernel call statement [0, -1]*/
{
if (getenv("IPMACC_VERBOSE")) printf("IPMACC: Launching kernel 0 > gridDim: %d\tblockDim: %d\n",(((abs((int)((NI))-(0+0)))/(1)))/256+(((((abs((int)((NI))-(0+0)))/(1)))%(256))==0?0:1),256);
__generated_kernel_region_0<<<(((abs((int)((NI))-(0+0)))/(1)))/256+(((((abs((int)((NI))-(0+0)))/(1)))%(256))==0?0:1),256>>>(
(DATA_TYPE *)acc_deviceptr((void*)A),
(DATA_TYPE *)acc_deviceptr((void*)B),
(DATA_TYPE *)acc_deviceptr((void*)C));
}
/* kernel call statement*/
if (getenv("IPMACC_VERBOSE")) printf("IPMACC: Synchronizing the region with host\n");
{
hipError_t err=hipDeviceSynchronize();
if(err!=hipSuccess){
printf("Kernel Launch Error! error code (%d)\n",err);
assert(0&&"Launch Failure!\n");}
}



}
	ipmacc_prompt((char*)"IPMACC: memory copyout A\n");
acc_copyout_and_keep((void*)A,(262143+0)*sizeof(DATA_TYPE ));
ipmacc_prompt((char*)"IPMACC: memory copyout B\n");
acc_copyout_and_keep((void*)B,(262143+0)*sizeof(DATA_TYPE ));
ipmacc_prompt((char*)"IPMACC: memory copyout C\n");
acc_copyout_and_keep((void*)C,(262143+0)*sizeof(DATA_TYPE ));



}

void init(DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *C, DATA_TYPE *C_OMP)
{
  int i, j;

  for (i = 0; i < NI; i++) {
    for (j = 0; j < NK; j++) {
      A [i * NK + j] = ((DATA_TYPE)i * j) / NI;
    }
  }

  for (i = 0; i < NK; i++) {
    for (j = 0; j < NJ; j++) {
      B [i * NJ + j] = ((DATA_TYPE)i * j + 1) / NJ;
    }
  }

  for (i = 0; i < NI; i++) {
    for (j = 0; j < NJ; j++) {
      C [i * NJ + j] = ((DATA_TYPE)i * j + 2) / NJ;
      C_OMP [i * NJ + j] = ((DATA_TYPE)i * j + 2) / NJ;
    }
  }
}

void compareResults(DATA_TYPE* C, DATA_TYPE* C_outputFromGpu)
{
  int i, j, fail;
  fail = 0;

  
  for (i = 0; i < NI; i++) {
    for (j = 0; j < NJ; j++) {
      if (percentDiff(C [i * NJ + j], C_outputFromGpu [i * NJ + j]) > PERCENT_DIFF_ERROR_THRESHOLD) {
        fail++;
      }
    }
  }

  
  printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}

int main(int argc, char *argv[])
{
  double t_start, t_end;

  DATA_TYPE* A;
  DATA_TYPE* B;
  DATA_TYPE* C;
  DATA_TYPE* C_outputFromGpu;

  A = (DATA_TYPE*)malloc(NI * NK * sizeof(DATA_TYPE));
  B = (DATA_TYPE*)malloc(NK * NJ * sizeof(DATA_TYPE));
  C = (DATA_TYPE*)malloc(NI * NJ * sizeof(DATA_TYPE));
  C_outputFromGpu = (DATA_TYPE*)malloc(NI * NJ * sizeof(DATA_TYPE));

  fprintf(stdout, "<< Matrix-multiply C=alpha.A.B+beta.C >>\n");

  init(A, B, C, C_outputFromGpu);

  t_start = rtclock();
  GPU__gemm(A, B, C_outputFromGpu);
  t_end = rtclock();
  fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);

  t_start = rtclock();
  CPU__gemm(A, B, C);
  t_end = rtclock();
  fprintf(stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);

  compareResults(C, C_outputFromGpu);

  free(A);
  free(B);
  free(C);
  free(C_outputFromGpu);

  return 0;
}



 __global__ void __generated_kernel_region_0(DATA_TYPE * A,DATA_TYPE * B,DATA_TYPE * C){
int __kernel_getuid_x=threadIdx.x+blockIdx.x*blockDim.x;
int __kernel_getuid_y=threadIdx.y+blockIdx.y*blockDim.y;
int __kernel_getuid_z=threadIdx.z+blockIdx.z*blockDim.z;
int  i;
int  k;
int  j;
{
{
{
 i=0+(__kernel_getuid_x);
if( i < NI)
{
for(j = 0; j < NJ; j++)
{
      C [i * NJ + j] *= BETA;
for(k = 0; k < NK; ++k)
{
        C [i * NJ + j] += ALPHA * A [i * NK + k] * B [k * NJ + j];
      }
}
}

}
}
}
//append writeback of scalar variables
}

